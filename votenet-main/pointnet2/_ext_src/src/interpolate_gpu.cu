#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// 
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// CUDA kernel to find three nearest neighbors
__global__ void three_nn_kernel(int b, int n, int m,
                                const float *__restrict__ unknown,
                                const float *__restrict__ known,
                                float *__restrict__ dist2,
                                int *__restrict__ idx) {
  // Implementation details omitted for brevity
}

void three_nn_kernel_wrapper(int b, int n, int m, const float *unknown,
                             const float *known, float *dist2, int *idx) {
  // CUDA stream management and kernel launch
}

// CUDA kernel for three-interpolation
__global__ void three_interpolate_kernel(int b, int c, int m, int n,
                                         const float *__restrict__ points,
                                         const int *__restrict__ idx,
                                         const float *__restrict__ weight,
                                         float *__restrict__ out) {
  // Implementation details omitted for brevity
}

void three_interpolate_kernel_wrapper(int b, int c, int m, int n,
                                      const float *points, const int *idx,
                                      const float *weight, float *out) {
  // CUDA stream management and kernel launch
}

// CUDA kernel for gradient computation in three-interpolation
__global__ void three_interpolate_grad_kernel(
    int b, int c, int n, int m, const float *__restrict__ grad_out,
    const int *__restrict__ idx, const float *__restrict__ weight,
    float *__restrict__ grad_points) {
  // Implementation details omitted for brevity
}

void three_interpolate_grad_kernel_wrapper(int b, int c, int n, int m,
                                           const float *grad_out,
                                           const int *idx, const float *weight,
                                           float *grad_points) {
  // CUDA stream management and kernel launch
}

